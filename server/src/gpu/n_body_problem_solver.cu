#include "hip/hip_runtime.h"
#include "server/gpu/n_body_problem_solver.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace n_body_problem_solver {

inline constexpr float SOFTENING_FACTOR = 0.001;
inline constexpr int NUM_THREADS = 256;

__global__ void one_step_euler_n_body_problem(float3* positions, float3* velocities, float* weights, float dt, float G, int n) {
    const int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        float acceleration_x = 0.0; 
        float acceleration_y = 0.0; 
        float acceleration_z = 0.0;

        for (int tile = 0; tile < gridDim.x; ++tile) {
            __shared__ float3 shared_positions[NUM_THREADS];
            __shared__ float shared_weights[NUM_THREADS];
            shared_positions[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];
            shared_weights[threadIdx.x] = weights[tile * blockDim.x + threadIdx.x];
            __syncthreads();

            for (int j = 0; j < NUM_THREADS; ++j) {
            float dx = shared_positions[j].x - positions[index].x;
            float dy = shared_positions[j].y - positions[index].y;
            float dz = shared_positions[j].z - positions[index].z;
            float dist_square = dx*dx + dy*dy + dz*dz + SOFTENING_FACTOR * SOFTENING_FACTOR;
            float inverse_dist = rsqrtf(dist_square);
            float inverse_dist_power_3 = inverse_dist * inverse_dist * inverse_dist;

            acceleration_x += dx * inverse_dist_power_3 * G * shared_weights[j]; 
            acceleration_y += dy * inverse_dist_power_3 * G * shared_weights[j]; 
            acceleration_z += dz * inverse_dist_power_3 * G * shared_weights[j];
            }
            __syncthreads();
        }

        velocities[index].x += dt * acceleration_x; 
        velocities[index].y += dt * acceleration_y; 
        velocities[index].z += dt * acceleration_z;

        positions[index].x += dt * velocities[index].x;
        positions[index].y += dt * velocities[index].y;
        positions[index].z += dt * velocities[index].z;
    }
}


SolutionNBodyProblem gpu_solve_n_body_problem(const NBodyProblemData& data) {
    float3* positions = new float3[data.num_bodies];
    float3* velocities = new float3[data.num_bodies];
    float* weigths = new float[data.num_bodies];

    for (int i = 0; i < data.num_bodies; ++i) {
        positions[i].x = data.bodies[i].position[0];
        positions[i].y = data.bodies[i].position[1];
        positions[i].z = data.bodies[i].position[2];

        velocities[i].x = data.bodies[i].vecocities[0];
        velocities[i].y = data.bodies[i].vecocities[1];
        velocities[i].z = data.bodies[i].vecocities[2];

        weigths[i] = data.bodies[i].weight;
    }

    SolutionNBodyProblem solution;
    solution.iterations.reserve(data.num_iterations);

    const int num_blocks = (data.num_bodies + NUM_THREADS - 1) / NUM_THREADS;

    float3* gpu_positions;
    float3* gpu_velocities;
    float* gpu_weights;

    hipMalloc((void**)&gpu_positions, sizeof(float3) * data.num_bodies);
    hipMalloc((void**)&gpu_velocities, sizeof(float3) * data.num_bodies);
    hipMalloc((void**)&gpu_weights, sizeof(float) * data.num_bodies);
    hipMemcpy(gpu_weights, weigths, sizeof(float) * data.num_bodies, hipMemcpyHostToDevice);

    for (int num_iteration = 0; num_iteration < data.num_iterations; ++num_iteration) {
        hipMemcpy(gpu_positions, positions, sizeof(float3) * data.num_bodies, hipMemcpyHostToDevice);
        hipMemcpy(gpu_velocities, velocities, sizeof(float3) * data.num_bodies, hipMemcpyHostToDevice);

        one_step_euler_n_body_problem<<<num_blocks, NUM_THREADS>>>(gpu_positions, gpu_velocities, gpu_weights, data.dt, data.G, data.num_bodies);

        hipMemcpy(positions, gpu_positions, sizeof(float3) * data.num_bodies, hipMemcpyDeviceToHost);
        hipMemcpy(velocities, gpu_velocities, sizeof(float3) * data.num_bodies, hipMemcpyDeviceToHost);

        std::vector<Body> bodies;
        bodies.reserve(data.num_bodies);

        for (int body_index = 0; body_index < data.num_bodies; ++body_index) {
            bodies.push_back(
                Body{
                    .name = data.bodies[body_index].name,
                    .position = {positions[body_index].x, positions[body_index].y, positions[body_index].z},
                    .vecocities = {velocities[body_index].x, velocities[body_index].y, velocities[body_index].z},
                    .weight = weigths[body_index]
                }
            );
        }

        solution.iterations.push_back(
            OneIterSolutionNBodyProblem{
                .num_iteration=num_iteration,
                .bodies=bodies
            }
        );
    }

    hipFree(gpu_positions);
    hipFree(gpu_velocities);
    hipFree(gpu_weights);

    delete[] positions;
    delete[] velocities;
    delete[] weigths;
    return solution;
}


} // namespace n_body_problem_solver
